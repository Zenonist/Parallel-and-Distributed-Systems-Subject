
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256 // size of vectors
#define T 64 //number of threads per block

__global__ void vecAdd(int *A){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < N)
        A[i] = i;
}

int main(int argc, char *argv[]){
    srand(1234);
    int i;
    int size = N * sizeof ( int);
    int a[N], *devA;
    hipMalloc( (void**)&devA, size);
    
    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    
    vecAdd<<<4,T>>>(devA);
    
    hipMemcpy( a, devA, size, hipMemcpyDeviceToHost);
    
    hipFree( devA);
    
    for (i = 0; i < N; i++){
        printf("%d ",a[i]);
    }
    printf("\n");
}