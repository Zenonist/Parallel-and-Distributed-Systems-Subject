
#include <hip/hip_runtime.h>
#include <stdio.h>
//If Width / TILE_WIDTH != integer
#define Width 35
#define TILE_WIDTH 14

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int ncols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    //Pvalue is used to store the element of the output matrix
    //that is computed by the threads
    
    float Pvalue = 0;
    if (row < ncols && col < ncols){
        for (int k = 0; k < ncols; ++k){
            float Melement = Md[row*ncols+k];
            float Nelement = Nd[k*ncols+col];
            Pvalue += Melement * Nelement;
        }
        Pd[row*ncols+col] = Pvalue;
    }
}

int main (int argc, char *argv[]){
    int i,j;
    int size = Width * Width * sizeof(float);
    float M[Width][Width],N[Width][Width],P[Width][Width];
    float* Md,*Nd,*Pd;
    
    for (i = 0; i < Width; i++){
        for (j = 0; j < Width; j++){
            M[i][j] = 1;
            N[i][j] = 2;
        }
    }
    
    hipMalloc( (void**)&Md, size);
    hipMalloc( (void**)&Nd, size);
    hipMalloc( (void**)&Pd, size);
    
    hipMemcpy( Md, M, size, hipMemcpyHostToDevice);
    hipMemcpy( Nd, N, size, hipMemcpyHostToDevice);
    
    //Calculate the block size
    int blocksize = (Width + TILE_WIDTH - 1) / TILE_WIDTH;
    
    //Setup the execution configuration
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(blocksize, blocksize);
    
    //Launch the device computation num_threads
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width );
    
    //Read P from the device 
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
    
    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
    
    for (i = 0; i < Width; i++){
        for (j = 0; j < Width; j++){
            printf("%.2f ",P[i][j]);
        }
        printf("\n");
    }
}