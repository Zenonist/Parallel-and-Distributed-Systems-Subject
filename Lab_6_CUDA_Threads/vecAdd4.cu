
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 578 // size of vectors
#define T 275 //number of threads per block
//Need to show array of a and b
__global__ void vecAdd(int *A ,int *B,int *C){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < N)
        C[i] = A[i] + B[i];
}

int main(int argc, char *argv[]){
    int blocks = (N + T - 1) / T;
    srand(1234);
    int i;
    int size = N * sizeof ( int);
    int a[N], b[N], c[N], *devA, *devB, *devC;
    for (i = 0;i < N; i++){
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }
    hipMalloc( (void**)&devA, size);
    hipMalloc( (void**)&devB, size);
    hipMalloc( (void**)&devC, size);
    
    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    hipMemcpy( devB, b, size, hipMemcpyHostToDevice);
    
    vecAdd<<<blocks,T>>>(devA,devB,devC);
    
    hipMemcpy( c, devC, size, hipMemcpyDeviceToHost);
    
    hipFree( devA);
    hipFree( devB);
    hipFree( devC);
    
    for (i = 0; i < N; i++){
        printf("%d ",c[i]);
    }
    printf("\n");
}