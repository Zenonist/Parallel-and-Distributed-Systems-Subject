
#include <hip/hip_runtime.h>
#include <stdio.h>

#define Width 32 // size of Width x Width matrix
#define TILE_WIDTH 16

__global__ void MatrixMulKernel (float* Md, float* Nd, float* Pd){
    __shared__ float shared_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float shared_B[TILE_WIDTH][TILE_WIDTH];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    //Pvalue is used to store the element of the output matrix
    // that is computed by the thread
    
    float Pvalue = 0;
    for (int k = 0; k < Width/TILE_WIDTH; k++){
        // y = row , x = col
        shared_A[threadIdx.y][threadIdx.x] = Md[row*Width+(k * TILE_WIDTH + threadIdx.x)];
        shared_B[threadIdx.y][threadIdx.x] = Nd[(k * TILE_WIDTH + threadIdx.y)*Width+col];
        __syncthreads(); // similar to barrier
        
        for (int x = 0; x < TILE_WIDTH; x++){
            Pvalue += shared_A[threadIdx.y][x] * shared_B[x][threadIdx.x];
        }
        __syncthreads(); // similar to barrier
    }
    Pd[row*Width+col] = Pvalue;
}

int main (int argc, char *argv[]){
    int i,j;
    int size = Width * Width * sizeof(float);
    float M[Width][Width], N[Width][Width], P[Width][Width];
    float* Md, *Nd, *Pd;
    
    for (i = 0; i < Width; i++){
        for (j = 0; j < Width; j++){
            M[i][j] = 1; N[i][j] = 2;
        }
    }
    
    hipMalloc( (void**)&Md, size);
    hipMalloc( (void**)&Nd, size);
    hipMalloc( (void**)&Pd, size);
    
    hipMemcpy( Md, M, size, hipMemcpyHostToDevice);
    hipMemcpy( Nd, N, size, hipMemcpyHostToDevice);
    
    //Setup the execution configuration
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(Width/TILE_WIDTH, Width/TILE_WIDTH);
    
    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
    
    //Read P from the device
    hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost);
    
    //Free device matrices
    hipFree( Md);
    hipFree( Nd);
    hipFree( Pd);
    
    for (i = 0; i < Width; i++){
        for (j = 0; j < Width; j++){
            printf("%.2f ",P[i][j]);
        }
        printf("\n");
    }
}