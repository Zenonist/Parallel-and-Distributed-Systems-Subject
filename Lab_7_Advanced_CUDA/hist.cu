
#include <hip/hip_runtime.h>
#include <stdio.h>
#define n 1024
#define NUMTHREADS 256
__global__ void histogram_kernel(unsigned int *data, unsigned int *bin) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        atomicAdd(&(bin[data[i]]), 1);
    }
}
int main(int argc, char *argv[]) {
    int i;
    int size = n * sizeof(int);
    unsigned int a[n];
    unsigned int bin[10];
    unsigned int *dA, *dBin;
    for (i = 0; i < n; i++) {
        a[i] = i % 10;
    }
    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&dBin, 10 * sizeof(int));
    
    hipMemcpy(dA, a, size, hipMemcpyHostToDevice);
    
    hipMemset(dBin, 0, 10 * sizeof(int));
    
    int nblocks = (n + NUMTHREADS - 1) / NUMTHREADS;
    
    histogram_kernel<<<nblocks, NUMTHREADS>>>(dA, dBin);
    
    hipMemcpy(bin, dBin, 10 * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dA);
    hipFree(dBin);
    
    int count = 0;
    for (i = 0; i < 10; i++) {
        printf("Freq %d = %d\n", i, bin[i]);
        count = count + bin[i];
    }
    printf("#elements = %d\n", count);
}