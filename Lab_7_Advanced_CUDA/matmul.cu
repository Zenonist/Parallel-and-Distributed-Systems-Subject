
#include <hip/hip_runtime.h>
#include <stdio.h>

#define Width 32 // size of Width x Width matrix
#define TILE_WIDTH 16

__global__ void MatrixMulKernel (float* Md, float* Nd, float* Pd, int ncols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    //Pvalue is used to store the element of the output matrix
    // that is computed by the thread
    
    float Pvalue = 0;
    for (int k = 0; k < ncols; ++k){
        float Melement = Md[row*ncols+k];
        float Nelement = Nd[k*ncols+col];
        Pvalue += Melement * Nelement;
    }
    Pd[row*ncols+col] = Pvalue;
}

int main (int argc, char *argv[]){
    int i,j;
    int size = Width * Width * sizeof(float);
    float M[Width][Width], N[Width][Width], P[Width][Width];
    float* Md, *Nd, *Pd;
    
    for (i = 0; i < Width; i++){
        for (j = 0; j < Width; j++){
            M[i][j] = 1; N[i][j] = 2;
        }
    }
    
    hipMalloc( (void**)&Md, size);
    hipMalloc( (void**)&Nd, size);
    hipMalloc( (void**)&Pd, size);
    
    hipMemcpy( Md, M, size, hipMemcpyHostToDevice);
    hipMemcpy( Nd, N, size, hipMemcpyHostToDevice);
    
    //Setup the execution configuration
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(Width/TILE_WIDTH, Width/TILE_WIDTH);
    
    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);
    
    //Read P from the device
    hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost);
    
    //Free device matrices
    hipFree( Md);
    hipFree( Nd);
    hipFree( Pd);
    
    for (i = 0; i < Width; i++){
        for (j = 0; j < Width; j++){
            printf("%.2f ",P[i][j]);
        }
        printf("\n");
    }
}