
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 2250
#define T 512

__global__ void vecReverse(int *a, int *b, int *c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){
        if (i % 2 == 0){
            c[i] = a[i] + b[i];
        }else{
            c[i] = a[i] - b[i];
        }
    }
}

int main(int argc, char *argv[]){
    int size = N * sizeof(int);
    int a[N], b[N], c[N], *devA, *devB, *devC;
    int blocks;
    if (N % T != 0){
        blocks =(N+T-1) / T;
    }else{
        blocks = N/T;
    }
        
    srand(1234);
    for (int i = 0; i < N; i++){
        a[i] = rand() % 1000;
        b[i] = rand() % 1000;
    }
    
    hipMalloc((void**)&devA, size);
    hipMalloc((void**)&devB, size);
    hipMalloc((void**)&devC, size);
    
    hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(devB, b, size, hipMemcpyHostToDevice);
    
    vecReverse<<<blocks,T>>>(devA,devB,devC);
    
    hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);
    
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    
    for (int i = 0; i < N; i++){
        printf("%d ",c[i]);
    }
    printf("\n");
}