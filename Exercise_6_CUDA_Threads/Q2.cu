
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2250
#define T 512

__global__ void vecReverse(int *a, int *b){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){
        b[i] = a[N - i - 1];
    }
}

int main(int argc, char *argv[]){
    int size = N * sizeof(int);
    int a[N], b[N], *devA, *devB;
    int blocks;
    if (N % T != 0){
        blocks =(N+T-1) / T;
    }else{
        blocks = N/T;
    }
    
    for (int i = 0; i < N; i++){
        a[i] = i;
    }
    
    hipMalloc((void**)&devA, size);
    hipMalloc((void**)&devB, size);
    
    hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
    
    vecReverse<<<blocks,T>>>(devA,devB);
    
    hipMemcpy(b, devB, size, hipMemcpyDeviceToHost);
    
    hipFree(devA);
    hipFree(devB);
    
    for (int i = 0; i < N; i++){
        printf("%d ",b[i]);
    }
    printf("\n");
}