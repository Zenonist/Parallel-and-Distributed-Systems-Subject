#include "hip/hip_runtime.h"
#include <stdio.h>

#define Width 8
#define T 2

__global__ void vecTranspose(int *a, int *b, int width) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    b[Col * width + Row] = a[Row * width + Col];
}

int main(){
    int size = Width * Width * sizeof(int);
    int a[Width][Width], b[Width][Width], *devA, * devB;
    srand(1234);
    printf("Original A array\n");
    for (int i = 0; i < Width; i++){
        for (int j = 0; j < Width; j++){
            a[i][j] = rand() % 1000;
            printf("%d ",a[i][j]);
        }
        printf("\n");
    }
    
    hipMalloc((void**)&devA, size);
    hipMalloc((void**)&devB, size);
    
    hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(T, T);
    dim3 dimGrid(Width/T, Width/T);
    
    vecTranspose<<<dimGrid, dimBlock>>>(devA, devB, Width, size);
    
    hipMemcpy(b, devB, size, hipMemcpyDeviceToHost);
    
    hipFree(devA);
    hipFree(devB);
    printf("\nNew A array \n");
    for (int i = 0; i < Width; i++){
        for (int j = 0; j < Width; j++){
            printf("%d ",b[i][j]);
        }
        printf("\n");
    }
}