
#include <hip/hip_runtime.h>
#include <stdio.h>

#define Width 16
#define T 4

__global__ void vecTranspose(int *a, int *b, int width , int size) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int k = 0; k < width; k++){
        //int index = Row * width + Col;
        b[Row * width + k] = a[k * width + Col];
        //printf("%d %d\n",b[Row * width + k],a[k * width + Col]);
        //printf("%d %d %d %d\n",k * width + Col,Row * width + k,k,index);
    }
}

int main(){
    int size = Width * Width * sizeof(int);
    int a[Width][Width], b[Width][Width], *devA, * devB;
    srand(1234);
    printf("Original A array\n");
    for (int i = 0; i < Width; i++){
        for (int j = 0; j < Width; j++){
            a[i][j] = rand() % 1000;
            printf("%d ",a[i][j]);
        }
        printf("\n");
    }
    
    hipMalloc((void**)&devA, size);
    hipMalloc((void**)&devB, size);
    
    hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(T, T);
    dim3 dimGrid(Width/T, Width/T);
    
    vecTranspose<<<dimGrid, dimBlock>>>(devA, devB, Width, size);
    
    hipMemcpy(b, devB, size, hipMemcpyDeviceToHost);
    
    hipFree(devA);
    hipFree(devB);
    printf("\nNew A array \n");
    for (int i = 0; i < Width; i++){
        for (int j = 0; j < Width; j++){
            printf("%d ",b[i][j]);
        }
        printf("\n");
    }
}