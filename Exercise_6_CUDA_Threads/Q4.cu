
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 100
#define T 20

__global__ void MatrixIncrement (int *a){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * N + x;
    a[index] = a[index] + 1;
}

int main(int argc, char *argv[]){
    int size = N * N * sizeof(int);
    int a[N][N] , *devA;
    srand(1234);
    
    printf("Original A array\n");
    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            a[i][j] = rand() % 1000;
            printf("%d ",a[i][j]);
        }
        printf("\n");
    }
    
    hipMalloc((void**)&devA, size);
    
    hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(T,T);
    dim3 dimGrid(N/dimBlock.x,N/dimBlock.y);
    
    MatrixIncrement<<<dimGrid,dimBlock>>>(devA);
    
    hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
    
    hipFree(devA);
    
    printf("\nNew A array \n");
    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            printf("%d ",a[i][j]);
        }
        printf("\n");
    }
}