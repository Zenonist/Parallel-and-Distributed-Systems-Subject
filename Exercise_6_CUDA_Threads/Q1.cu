
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1250
#define T 250

__global__ void vecAssign(int *a){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){
        a[i] = i * 2;
    }
}

int main(int argc, char *argv[]){
    int size = N * sizeof(int);
    int a[N], *devA;
    int blocks;
    //Compute the blocks in case that N % T != 0
    if (N % T != 0){
        blocks =(N+T-1) / T;
    }else{
        blocks = N/T;
    }
    
    hipMalloc( (void**)&devA, size);
    
    hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
    
    vecAssign<<<blocks,T>>>(devA);
    
    hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
    
    hipFree(devA);
    
    for (int i = 0; i < N; i++){
        printf("%d ",a[i]);
    }
    printf("\n");
}