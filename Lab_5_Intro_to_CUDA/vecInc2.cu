
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 1000
#define T 256
__global__ void vecInc(int *A,int *newA){
    int i;
    for (i = threadIdx.x;i < N;i = i + T){
	newA[i] = A[i] + 1;
    }
}
int main (int argc, char *argv[]){
    int i;
    int size = N * sizeof ( int);
    int a[N], new_a[N], *devA, *dev_newA;
    printf("Original A array\n");
    for (i = 0; i < N; i++){
        a[i] = rand() % 100;
	printf("%d ",a[i]);
    }
    hipMalloc( (void**)&devA, size);
    hipMalloc( (void**)&dev_newA, size);

    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);

    vecInc<<<1, T>>>(devA, dev_newA);
    hipMemcpy( new_a, dev_newA, size, hipMemcpyDeviceToHost);
    hipFree( devA);
    hipFree( dev_newA);
    printf("\nNew A array \n");
    for (i= 0; i < N; i++){
        printf("%d ",new_a[i]);
    }
    printf("\n");
}