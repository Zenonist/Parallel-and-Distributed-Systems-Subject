
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1314
#define T 256

__global__ void vecMult(int *A ,int *B){
    int i = 0;
    for (i = threadIdx.x; i < N;i = i + T)
        B[i] = A[i] * 2;
}
int main (int argc, char *argv[]){
    int i = 0;
    int size = N * sizeof (int);
    int a[N], b[N], *devA, *devB;
    for (i = 0; i < N; i++){
        a[i] = i;
    }
    hipMalloc( (void**)&devA,size);
    hipMalloc( (void**)&devB,size);
    
    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    
    vecMult<<<1,T>>>(devA,devB);
    
    hipMemcpy( b, devB, size, hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    for (i = 0;i < N; i++){
        printf("%d ",b[i]);
    }
    printf("\n");
}