
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256
#define T 256
__global__ void reverseArray(int *A ,int *B){
    int i = threadIdx.x;
    B[i] = A[(N - 1) - i];
}
int main (int argc, char *argv[]){
    int i;
    int size = N * sizeof (int);
    int a[N], b[N], *devA, *devB;
    printf("Original A array\n");
    for (i = 0; i < N; i++){
        a[i] = i;
	printf("%d ",a[i]);
    }
    hipMalloc( (void**)&devA,size);
    hipMalloc( (void**)&devB,size);
    
    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    
    reverseArray<<<1,T>>>(devA,devB);
    
    hipMemcpy( b, devB, size, hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    printf("\nNew A array \n");
    for (i = 0;i < N; i++){
        printf("%d ",b[i]);
    }
    printf("\n");
}