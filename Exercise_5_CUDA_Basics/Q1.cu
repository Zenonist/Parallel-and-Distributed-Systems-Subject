
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256
#define T 256

__global__ void vecMult(int *A ,int *B){
    int i = threadIdx.x;
    B[i] = A[i] * 2;
}
int main (int argc, char *argv[]){
    int i;
    int size = N * sizeof (int);
    int a[N], b[N], *devA, *devB;
    for (i = 0; i < N; i++){
        a[i] = i;
    }
    hipMalloc( (void**)&devA,size);
    hipMalloc( (void**)&devB,size);
    
    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    
    vecMult<<<1,T>>>(devA,devB);
    
    hipMemcpy( b, devB, size, hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    for (i = 0;i < N; i++){
        printf("%d ",b[i]);
    }
    printf("\n");
}