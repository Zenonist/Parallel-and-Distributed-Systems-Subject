
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1024
#define T 256

__global__ void FindFreq(int *devA, int target , int *result){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N){
        //printf("%d-%d\n",devA[index],target);
        //printf((int) target == (int) devA[index] ? "true\n" : "false\n");
        if (target == devA[index]){
            atomicAdd(&result[0],1);
        }
    }
    __syncthreads();
}

int main (int argc, char *argv[]){
    int size = N * sizeof (int);
    int A[N] , *devA , *devResult;
    int targetnumber;
    int result[1]; //It should be right because we need to send array[pointer] to Device
    printf("Enter the target number to find the frequency (integer only): ");
    scanf("%d", &targetnumber);
    for (int x = 0; x < N; x++){
        A[x] = rand () % 100;
        printf("%d ",A[x]);
    }
    
    hipMalloc((void**)&devA, size);
    hipMalloc((void**)&devResult, 1 * sizeof(int));
    
    hipMemcpy(devA, A, size, hipMemcpyHostToDevice);
    
    hipMemset(devResult, 0, 1 * sizeof(int));
    
    dim3 dimBlock(T);
    dim3 dimGrid(N/T);
    
    FindFreq<<<dimGrid, dimBlock>>>(devA, targetnumber , devResult);
    
    hipMemcpy(result, devResult, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devResult);
    
    printf("\nThe frequency of %d in array: %d",targetnumber, result[0]);
}