
#include <hip/hip_runtime.h>
#include <stdio.h>
#define Width 16
#define TILE_WIDTH 4

__global__ void MatrixCalSum(int *devA, int *devB){
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    atomicAdd(&devB[row], devA[Width*col+row]);
    atomicAdd(&devB[col+8], devA[Width*col+row]);
    

}
int main(int argc, char *argv[]){
    int size = Width * Width * sizeof(int);
    int A[Width][Width] , Result[Width], *devA , *devB;
    for (int x = 0;x < Width; x++){
        for (int y = 0; y < Width; y++){
            A[x][y] = rand() % 1000;
            printf("%d ", A[x][y]);
        }
        printf("\n");
    }
    hipMalloc((void **)&devA,size);
    hipMalloc((void **)&devB, Width * sizeof(int));
    
    hipMemcpy(devA, A, size, hipMemcpyHostToDevice);
    
    hipMemset(devB, 0, Width * sizeof(int));
    
    dim3 dimBlock(TILE_WIDTH);
    dim3 dimGrid(Width/TILE_WIDTH);
    
    MatrixCalSum<<<dimGrid, dimBlock>>>(devA, devB);
    
    hipMemcpy(Result, devB, Width * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(devA);
    hipFree(devB);
    
    /*for (int x = 0; x < Width + 1; x++){
        for (int y = 0; y < Width; y++){
            printf("%d ",A[x][y]);
            if (y == Width - 1 && x != Width){
                printf("| %d\n",Result[y]);
            }
            if (x == Width){
                printf("%d ",Result[8 + y]);
            }
        }
    }*/
    for (int x = 0; x < 16;x++){
        printf("%d ",Result[x]);
    }
}