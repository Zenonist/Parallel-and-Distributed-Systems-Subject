
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 64
#define T 16

__global__ void RankSort (int *devA, int *devB){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int temp = devA[index];
    
    int count = 0;
    for (int x = 0; x < N; x++){
        // We need index > x because we want to move the duplicate number to next index.
        if (temp > devA[x] || temp == devA[x] && index > x){
            count++;
        }
    }
    devB[count] = temp;
}

int main (int argc, char *argv[]){
    int a[N], *devA, *devB;
    int size = N * sizeof (int);
    printf("Orgianal A array \n");
    for (int x= 0; x < N; x++){
        a[x] = rand () % 100;
        printf ("%d ",a[x]);
    }
    printf("\n");
    hipMalloc( (void**)&devA, size);
    hipMalloc( (void**)&devB, size);
    
    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(T);
    dim3 dimGrid(N/T);
    
    RankSort<<<dimGrid, dimBlock>>>(devA,devB);
    
    hipMemcpy(a, devB, size, hipMemcpyDeviceToHost);
    
    hipFree(devA);
    hipFree(devB);
    
    printf("\nnew A array\n");
    for (int x = 0; x < N; x++){
        printf("%d ",a[x]);
    }
    printf("\n");
}